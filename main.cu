#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))


int main (int argc, char *argv[])
{

  FILE *inA;
  FILE *inB;
  int bufferINT;
  float bufferFLOAT;

  int m; //6
  int n; //4
  int k; //5

  inA = fopen(argv[1], "r");
  inB = fopen(argv[2], "r");

  if (inA == NULL || inB == NULL) {
    printf("Couldn’t open file for reading. \n");
    return 0;
  }

  fread(&bufferINT, sizeof(unsigned int), 1, inA);
  int a_row = bufferINT;
  fread(&bufferINT, sizeof(unsigned int), 1, inA);
  int a_col = bufferINT;

  fread(&bufferINT, sizeof(unsigned int), 1, inB);
  int b_row = bufferINT;
  fread(&bufferINT, sizeof(unsigned int), 1, inB);
  int b_col = bufferINT;

  if(a_col != b_row)
  {
    printf("Matrices can't be multiplied");
    return 0;
  }
  else
  {
    m = a_row;
    n = b_col;
    k = a_col;
  }


  hipError_t cudaStat;                                        // cudaMalloc status
  hipblasStatus_t stat;                                         // CUBLAS functions status
  hipblasHandle_t handle;                                       // CUBLAS context
  int i,j;                                                      // i-row index ,j- column index
  float * a;                                                    // mxk matrix a on the host
  float * b;                                                    // kxn matrix b on the host
  float * c;                                                    // mxn matrix c on the host
  a=(float*) malloc (m*k* sizeof(float));                  // host memory for a
  b=(float*) malloc (k*n* sizeof(float));                  // host memory for b
  c=(float*) malloc (m*n* sizeof(float));                  // host memory for c


  // define an mxk matrix a column by column
   // a:
  for(j=0;j<k;j++){
    for(i=0;i<m;i++){
      fread(&bufferFLOAT, sizeof(float), 1, inA);
      a[IDX2C(i,j,m)]=bufferFLOAT;
    }
  }


/*  // print a row by row
  printf ("a:\n");
  for (i=0;i<m;i++){
    for (j=0;j<k;j++){
      printf ("%5.0f",a[IDX2C(i,j,m )]);
    }
    printf ("\n");
  }*/


  // define a kxn matrix b column by column
  for(j=0;j<n;j++){
    for(i=0;i<k;i++){
      fread(&bufferFLOAT, sizeof(float), 1, inB);
      b[IDX2C(i,j,k)]=bufferFLOAT;
    }
  }


/*  // print b row by row
  printf ("b:\n");
  for (i=0;i<k;i++){
    for (j=0;j<n;j++){
      printf ("%5.0f",b[IDX2C(i,j,k )]);
    }
    printf ("\n");
  }*/


  // define an mxn matrix c column by column
  int ind =0;
  for(j=0;j<n;j++){
    for(i=0;i<m;i++){
      c[IDX2C(i,j,m)]=(float)ind;
    }
  }


/*  // print c row by row
  printf ("c:\n");
    for (i=0;i<m;i++){
      for (j=0;j<n;j++){
        printf ("%5.0f",c[IDX2C(i,j,m)]);
      }
      printf ("\n");
  }*/

  fclose(inA);
  fclose(inB);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  // on the device
  float* d_a; // d_a - a on the device
  float* d_b; // d_b - b on the device
  float* d_c; // d_c - c on the device

  cudaStat = hipMalloc((void**)&d_a,m*k*sizeof(*a));
  // memory alloc for a

  cudaStat = hipMalloc((void**)&d_b,k*n*sizeof(*b));
  // memory alloc for b

  cudaStat = hipMalloc((void**)&d_c,m*n*sizeof(*c));
  // memory alloc for c

  stat = hipblasCreate(&handle); // initialize CUBLAS context


  // copy matrices from the host to the device
  stat = hipblasSetMatrix(m,k,sizeof(*a),a,m,d_a,m); //a -> d_a
  stat = hipblasSetMatrix(k,n,sizeof(*b),b,k,d_b,k); //b -> d_b
  stat = hipblasSetMatrix(m,n,sizeof(*c),c,m,d_c,m); //c -> d_c
  float al =1.0f; // al =1
  float bet =1.0f; // bet =1


  // matrix - matrix multiplication : d_c = al*d_a *d_b + bet *d_c
  // d_a -mxk matrix , d_b -kxn matrix , d_c -mxn matrix ;
  // al ,bet -scalars
  hipEventRecord(start);

  stat=hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,d_a,
  m,d_b,k,&bet,d_c,m);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float seconds = 0;
  float gflops = 0;
  float calc = n;
  hipEventElapsedTime(&seconds, start, stop);
  gflops = 2 * pow(calc, 3);
  gflops = gflops / seconds;

  printf("Time: %.6f \n", seconds);
  printf("GFLOP/s: %.5f \n", gflops);



  stat = hipblasGetMatrix (m,n,sizeof(*c),d_c ,m,c,m); // cp d_c - >c

  /*printf ("c after Sgemm :\n");
  for(i=0;i<m;i ++){
    for(j=0;j<n;j ++){
      printf ("%7.0f",c[ IDX2C (i,j,m )]); // print c after Sgemm
    }
    printf ("\n");
  }*/


  hipFree (d_a); // free device memory
  hipFree (d_b); // free device memory
  hipFree (d_c); // free device memory

  hipblasDestroy (handle); // destroy CUBLAS context

  free (a); // free host memory
  free (b); // free host memory
  free (c); // free host memory
  return EXIT_SUCCESS ;
}
